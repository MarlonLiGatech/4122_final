#include "hip/hip_runtime.h"
#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <cstring>
#include <string>
#include <sstream>
#include <thrust/host_vector.h>
#define T_P_B 1024

__global__ void heat2D(float *in, float *out, float *d_candles, int cells, float k, int width) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < cells) {
        float top = in[idx];
        float bot = in[idx];
        float left = in[idx];
        float right = in[idx];

        if (idx >= width)    //top row
            top = in[idx - width];
        if (idx < (cells - width)) //bottom row
            bot = in[idx + width];
        if (idx % width != 0)  //left column
            left = in[idx - 1];
        if ((idx + 1) % width != 0)  //right column
            right = in[idx + 1];
        if (d_candles[idx] >= 0)
            out[idx] = d_candles[idx];
        else
            out[idx] = in[idx] + k * (top + bot + right + left - 4 * in[idx]);
    }
}

int main(int argc, char *argv[]) {
    if (argc != 4)
    {
        std::cout << "Incorrect usage" << std::endl;
        return -1;
    }

    // declare vars
    InputImage imageObj(argv[2]);
    int length, totalLength;
    Complex *image = nullptr;

    // values from input file
    length = imageObj.get_width();
    totalLength = length * length;
    image = imageObj.get_image_data();



    
    //create stationary heating source grid
    int cells = width * height * depth;
    float candles[cells] = {};
    for(int i = 0; i < cells; ++i) {
        candles[i] = -1;
    }

    //input candle information
    int location_x, location_y, candle_width, candle_height;
    int location_z = 0, candle_depth = 1;
    float fixed_temperature;
    int ind;
    for (int i = 0; i < static_heat.size()/info_size; i++) {
        ind = i * info_size;
        location_x = static_heat[0 + ind];
        location_y = static_heat[1 + ind];
        if (!mode) {
            candle_width = static_heat[2 + ind];
            candle_height = static_heat[3 + ind];
            fixed_temperature = static_heat[4 + ind];
        } else {
            location_z = static_heat[2 + ind];
            candle_width = static_heat[3 + ind];
            candle_height = static_heat[4 + ind];
            candle_depth = static_heat[5 + ind];
            fixed_temperature = static_heat[6 + ind];
        }
        for (int z = location_z; z < location_z + candle_depth; z++) {
            for (int y = location_y; y < location_y + candle_height; y++) {
                for (int x = location_x; x < location_x + candle_width; x++) {
                    candles[x + y * width + z * width * height] = fixed_temperature;
                }
            }
        }
    }

    //cuda code
    float *d_gridA, *d_gridB, *d_candles;
    hipMalloc((void**)&d_gridA, cells * sizeof(float));
    hipMalloc((void**)&d_gridB, cells * sizeof(float));
    hipMalloc((void**)&d_candles, cells * sizeof(float));

    float gridA[cells], gridB[cells];

    //initialize starting grid
    for(int i = 0; i < cells; ++i)
        gridA[i] = starting_temp;

    //copy candles
    for (int i = 0; i < cells; i++) {
        if (candles[i] >= 0)
            gridA[i] = candles[i];
    }

    hipMemcpy(d_gridA, gridA, cells * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_candles, candles, cells * sizeof(float), hipMemcpyHostToDevice);

    float *in, *out;
    int i;
    for (i = 0; i < timestep; i++) {
        if (i % 2) {
            in = d_gridB;
            out = d_gridA;
        } else {
            in = d_gridA;
            out = d_gridB;
        }

        if (mode == 0)
            heat2D<<<(cells + T_P_B-1) / T_P_B, T_P_B>>>(in, out, d_candles, cells, k, width);
        else
            heat3D<<<(cells + T_P_B-1) / T_P_B, T_P_B>>>(in, out, d_candles, cells, k, width, height);
    }

    //set up answer
    float *answer;
    if (i % 2) {
        hipMemcpy(gridB, d_gridB, cells*sizeof(float), hipMemcpyDeviceToHost);
        answer = gridB;
    } else {
        hipMemcpy(gridA, d_gridA, cells*sizeof(float), hipMemcpyDeviceToHost);
        answer = gridA;
    }
    hipFree(d_gridA);
    hipFree(d_gridB);
    hipFree(d_candles);

    //print answers
    std::ofstream myfile;
    myfile.open("heatOutput.csv");
    for(int i = 0; i < cells; ++i) {
        myfile << answer[i];
        if ((((i + 1) % width) == 0) && i > 0 && i < (cells - 1))
            myfile << std::endl;
        else
            myfile << ",";
        if ((((i + 1) % (width * height)) == 0) && i < (cells - 1))
            myfile << std::endl;
    }
    myfile.close();

    return 0;
}